/*
 Name: Andrew Hahn
 Email: ahahn1@crimson.ua.edu
 Course Section: CS 581
 Homework #: 5
 To Compile: nvcc HW5.cu -o HW5
 To Run: ./HW5 5000 5000
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <sys/time.h>

#define DEAD 0
#define ALIVE 1

#define BLOCK_SIZE 16 // Adjust block size as needed

__global__ void evolve(int *current, int *next, int size) {
    int x = blockIdx.x * blockDim.x + threadIdx.x + 1; // +1 because of padding
    int y = blockIdx.y * blockDim.y + threadIdx.y + 1; // +1 because of padding

    int idx = y * (size + 2) + x;

    if (x <= size && y <= size) {
        int aliveNeighbors = 0;

        // Offsets for the 8 neighbors
        int offsets[] = {
            - (size + 2) - 1,  // Top-left
            - (size + 2),      // Top
            - (size + 2) + 1,  // Top-right
            -1,                // Left
            +1,                // Right
            (size + 2) - 1,    // Bottom-left
            (size + 2),        // Bottom
            (size + 2) + 1     // Bottom-right
        };

        // Loop over neighbors
        for (int i = 0; i < 8; i++) {
            int neighbor_idx = idx + offsets[i];
            aliveNeighbors += current[neighbor_idx];
        }

        // Apply the Game of Life rules
        if (current[idx] == ALIVE) {
            next[idx] = (aliveNeighbors == 2 || aliveNeighbors == 3) ? ALIVE : DEAD;
        } else {
            next[idx] = (aliveNeighbors == 3) ? ALIVE : DEAD;
        }
    }
}

void initializeBoard(int *board, int size) {
    // Seed the random number generator
    srand(52);

    int totalSize = (size + 2) * (size + 2);

    // Initialize all cells to DEAD
    for (int i = 0; i < totalSize; i++) {
        board[i] = DEAD;
    }

    // Randomly set cells to ALIVE or DEAD
    for (int y = 1; y <= size; y++) {
        int row = y * (size + 2);
        for (int x = 1; x <= size; x++) {
            int idx = row + x;
            board[idx] = (rand() % 2 == 0) ? DEAD : ALIVE;
        }
    }
}

void writeBoardToFile(int *board, int size, const char *filename) {
    FILE *fp = fopen(filename, "w");
    if (fp == NULL) {
        printf("Error opening file %s for writing\n", filename);
        return;
    }
    for (int y = 1; y <= size; y++) {
        int row = y * (size + 2);
        for (int x = 1; x <= size; x++) {
            fprintf(fp, board[row + x] == ALIVE ? "O " : ". ");
        }
        fprintf(fp, "\n");
    }
    fclose(fp);
}

int main(int argc, char *argv[]) {
    // Start timer
    struct timeval start, end;
    gettimeofday(&start, NULL);

    // Check if the correct number of arguments are given
    if (argc != 3) {
        printf("Usage: %s <size of board> <max generations>\n", argv[0]);
        return 1;
    }

    // Create variables
    int size = atoi(argv[1]);
    int maxGenerations = atoi(argv[2]);

    // Allocate host memory
    int *h_board1 = (int *)malloc((size + 2) * (size + 2) * sizeof(int));
    int *h_board2 = (int *)malloc((size + 2) * (size + 2) * sizeof(int));

    // Initialize boards
    initializeBoard(h_board1, size);
    initializeBoard(h_board2, size); // Initialize board2 to avoid uninitialized memory

    // Allocate device memory
    int *d_board1, *d_board2;
    hipMalloc((void **)&d_board1, (size + 2) * (size + 2) * sizeof(int));
    hipMalloc((void **)&d_board2, (size + 2) * (size + 2) * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_board1, h_board1, (size + 2) * (size + 2) * sizeof(int), hipMemcpyHostToDevice);

    int generation = 0;

    // Define block and grid sizes
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((size + BLOCK_SIZE - 1) / BLOCK_SIZE, (size + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Run the game
    while (generation < maxGenerations) {
        // Launch the kernel
        evolve<<<gridSize, blockSize>>>(d_board1, d_board2, size);

        // Swap the boards
        int *temp = d_board1;
        d_board1 = d_board2;
        d_board2 = temp;

        generation++;
    }

    // Copy final board back to host
    hipMemcpy(h_board1, d_board1, (size + 2) * (size + 2) * sizeof(int), hipMemcpyDeviceToHost);

    // Write final board to file
    writeBoardToFile(h_board1, size, "outputs/final_board.txt");

    // Free device memory
    hipFree(d_board1);
    hipFree(d_board2);

    // Free host memory
    free(h_board1);
    free(h_board2);

    // End timer and calculate total time
    gettimeofday(&end, NULL);
    double elapsed = (end.tv_sec - start.tv_sec) + ((end.tv_usec - start.tv_usec)/1e6);
    printf("Total time taken: %f seconds\n", elapsed);

    return 0;
}
